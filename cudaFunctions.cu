#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <string.h>
using namespace std;
#include "functions.h"
#include "cudaFunctions.h"


/// @brief Checking if the two letters are in the same group.
/// @param word The word in the group.
/// @param letter1 Letter from A-Z.
/// @param letter2 Letter from A-Z.
/// @return TRUE if find the two letter in the word or FALSE if not.
__device__ int isLettersInWordCuda(const char *word, char letter1, char letter2)
{
    int i = 0;
    int hasletter1 = FALSE, hasletter2 = FALSE;
    while (word[i] != '\0')
    {
        if (word[i] == letter1)
            hasletter1 = TRUE;
        else if (word[i] == letter2)
            hasletter2 = TRUE;
        i++;
    }
    if (hasletter1 == TRUE && hasletter2 == TRUE)
        return TRUE;

    return FALSE;
}

/// @brief  Checking if the two letters are in the same group.
/// @param group The group with words.
/// @param letter1 Letter from A-Z.
/// @param letter2 Letter from A-Z.
/// @param size Size of the group.
/// @return TRUE if find the two letter existing in the group or FALSE if not.
__device__ int isLettersInGroupCuda(const char **group, char letter1, char letter2, int size)
{
    for (int i = 0; i < size; i++)
    {
        if (isLettersInWordCuda(group[i], letter1, letter2) == TRUE)
            return TRUE;
    }
    return FALSE;
}


/// @brief  Fill the matrix with scores for every to letters by GPU.
/// @param devScoreMatrix Empty score matrix 26*26 from the GPU.
/// @param devWeight Weights for each coupling w[0] w[1] w[2] w[3] from the GPU.
__global__ void fillMatrix(int *devScoreMatrix, int *devWeight)
{
    const char *firstGroup[FIRST_GROUP_SIZE] = {"NDEQ", "NEQK", "STA", "MILV", "QHRK", "NHQK", "FYW", "HY", "MILF"};
    const char *secondGroup[SECOND_GROUP_SIZE] = {"SAG", "ATV", "CSA", "SGND", "STPA", "STNK", "NEQHRK", "NDEQHK", "SNDEQK", "HFY", "FVLIM"};

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < ENGLISH_LETTERS && j < ENGLISH_LETTERS)
    {
        char letter1 = i + 'A';
        char letter2 = j + 'A';
        if (letter1 == letter2)
            devScoreMatrix[i * ENGLISH_LETTERS + j] = devWeight[0];
        else if (isLettersInGroupCuda(firstGroup, letter1, letter2, FIRST_GROUP_SIZE))
            devScoreMatrix[i * ENGLISH_LETTERS + j] = -devWeight[1];
        else if (isLettersInGroupCuda(secondGroup, letter1, letter2, SECOND_GROUP_SIZE))
            devScoreMatrix[i * ENGLISH_LETTERS + j] = -devWeight[2];
        else
            devScoreMatrix[i * ENGLISH_LETTERS + j] = -devWeight[3];
    }
}

/**
 * Check if cuda status success.
 * @param int* pointer to array.
 * @return EXIT_SUCCESS if everything worked properly, EXIT_FAILURE else.
 */
int checkCudaMalloc(int *array)
{
    if (array == NULL)
    {
        printf("Cuda malloc failed!");
        hipFree(array);
        return EXIT_FAILURE;
    }

    return EXIT_SUCCESS;
}

/// @brief Fill the matrix with scores for every to letters.
/// @param scoreMatrix Empty score matrix 26*26 from the cpu.
/// @param weights Weights for each coupling w[0] w[1] w[2] w[3] from the cpu.
/// @return EXIT_SUCCESS or EXIT_FAILURE if the func fill the matrix with scores.
int fillSymbolsWeightsCuda(int *scoreMatrix, int *weights)
{
    int *devWeights = NULL, *devScoreMatrix = NULL;
    int threads = ENGLISH_LETTERS / 2;
    dim3 THREADS(threads, threads);
    dim3 BLOCKS(2, 2);

    hipMalloc((void **)&devScoreMatrix, ENGLISH_LETTERS * ENGLISH_LETTERS * sizeof(int));
    if (checkCudaMalloc(devScoreMatrix) == EXIT_FAILURE)
        return EXIT_FAILURE;
    hipMalloc((void **)&devWeights, NUM_OF_WEIGHTS * sizeof(int));
    if (checkCudaMalloc(devWeights) == EXIT_FAILURE)
        return EXIT_FAILURE;

    hipMemcpy(devScoreMatrix, scoreMatrix, ENGLISH_LETTERS * ENGLISH_LETTERS * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(devWeights, weights, NUM_OF_WEIGHTS * sizeof(int), hipMemcpyHostToDevice);

    fillMatrix<<<BLOCKS, THREADS>>>(devScoreMatrix, devWeights);
    hipDeviceSynchronize();

    hipMemcpy(scoreMatrix, devScoreMatrix, ENGLISH_LETTERS * ENGLISH_LETTERS * sizeof(int), hipMemcpyDeviceToHost);
    hipFree(devScoreMatrix);
    hipFree(devWeights);
    return EXIT_SUCCESS;
}